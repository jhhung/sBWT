#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <iostream>
#include <fstream>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <algorithm>
#include "cudaec.cpp"
#include <sys/time.h>
#include <unistd.h>
#include <string>
#include <sstream>
#include <vector>
#include <iterator>
#include <boost/archive/binary_iarchive.hpp>
#include <boost/archive/binary_oarchive.hpp>
#include <boost/serialization/string.hpp>
#include <boost/serialization/vector.hpp>
#include <boost/serialization/map.hpp>
#include <boost/lexical_cast.hpp>
#include <memory>

#include "my_genome_pre_handler.hpp"

typedef unsigned short Type;
typedef long long INTTYPE;
#define CARD_MEMORY_LIMIT 6442450944
// #define CARD_MEMORY_LIMIT 1048576
#define SUFFIXLEN 256 //###
//#define SUFFIXLEN 16 //###
#define TBLSTRIDE 32 //###
//#define TBLSTRIDE 2 //###
#define TYPEBITS 16
#define TYPEBYTES sizeof(Type)
#define ENCODEBITS 2
#define ENCODELEN (TYPEBITS/ENCODEBITS)
#define ENCODERATIO (8/ENCODEBITS)
#define MAXREADLEN 128
#define DIMB 16
#define LEN_HANDLE_CHAR 32
#define LEN_CHR 64

#define OVERNUM 1 

#define MAX_DIM_THD 1024
#define MAX_DIM_BLK 65536

//#define MYFILE 1

//#define IS_CREATE_TBL 1
//#define IS_FIND_READS 1

enum DNA_enum{A = 0, C, G, T, DOLLAR};
enum STRAND_opt_enum{FIND_N_STRAND = 0, FIND_P_STRAND, FIND_BOTH_STRAND};
enum SAM_FLAG{MAPPED = 0, REVERSE_COMPLEMENTED = 16};

struct Timer {
	char *topic;
	timeval start, end;
	
	Timer(char *itopic = NULL): topic(itopic) {
		fprintf(stderr, "[start: %s]\n", topic);
		gettimeofday(&start, NULL);
	}

	~Timer() {
		gettimeofday(&end, NULL);
		const float timeuse = 1000000 * (end.tv_sec - start.tv_sec) + end.tv_usec - start.tv_usec;
		fprintf(stderr, "[timer: %s] %f\n", topic, (timeuse / 1000000));
	}
};


template <class Value, class Compare>
__device__ __host__
int upper_bound(Value data[], int it_first, int it_last, const Value &value, Compare comp)
{
	int it;
	int count, step;
	count = it_last - it_first;

	while (count > 0)
	{
		it = it_first;
		step = count / 2;
		it = it + step;
		
		if (!comp(value, data[it]))
		{
			it_first = ++it;
			count -= (step + 1);
		}
		else
		{
			count = step;
		}
	}
	
	return it_first;
}


struct INTTYPEComp
{
	__host__ __device__
	bool operator() (const INTTYPE &a, const INTTYPE &b)
	{
		return (a < b);
	}
};


struct PositionChecker
{
	int _realSize;
	INTTYPE *chr_start_pos_key;
	INTTYPE *NPosLen_key;
	INTTYPE *NPosLen_val;
	int len_chr_start_pos;
	int len_NPosLen;

	__host__ __device__
	bool is_vaild(int len_read, int &lowerIter, int &position, bool &isRC, int strand)
	{
		if (position >= _realSize)
		{
			isRC = true;
			position = _realSize * 2 - position - len_read;
		}
		else
		{
			isRC = false;
		}

		if ((strand == 0 && !isRC) || (strand == 1 && isRC))
			return false;

		lowerIter = upper_bound(chr_start_pos_key, 0, len_chr_start_pos, (INTTYPE)position, INTTYPEComp());
		if (lowerIter > 0) lowerIter = lowerIter - 1;
		
		int lowerIter3 = upper_bound(chr_start_pos_key, 0, len_chr_start_pos, (INTTYPE)(position + len_read - 1), INTTYPEComp());
		if (lowerIter3 > 0) lowerIter3 = lowerIter3 - 1;

		if (lowerIter != lowerIter3)
			return false;

		int NLowerIter = upper_bound(NPosLen_key, 0, len_NPosLen, (INTTYPE)position, INTTYPEComp());
		if (NLowerIter > 0) NLowerIter = NLowerIter - 1;

		int NLowerIter3 = upper_bound(NPosLen_key, 0, len_NPosLen, (INTTYPE)(position + len_read - 1), INTTYPEComp());
		if (NLowerIter3 > 0) NLowerIter3 = NLowerIter3 - 1;

		if (NLowerIter != NLowerIter3)
			return false;

		position = position - chr_start_pos_key[lowerIter] + NPosLen_val[NLowerIter];

		return true;
	}
};


template <class T>
void archive_save(std::string filename, T &data) {
	typedef boost::archive::binary_oarchive oarchive;

	std::ofstream outf(filename.c_str());
	if (!outf.is_open()) {
		std::cerr << "oArchive file open failed\n";
	}

	oarchive oa(outf);
	oa << data;
	
	outf.close();
}

template <class T>
void archive_load(std::string filename, T &data) {
	typedef boost::archive::binary_iarchive iarchive;

	std::ifstream inf(filename.c_str(), std::ios::binary);
	if (!inf.is_open()) {
		std::cerr << "iArchive file open failed\n";
	}

	iarchive ia(inf);
	ia >> data;

	inf.close();
}


inline __host__ __device__ int get_token_id(char *token) {
	switch (*token) {
		case 'A':
			return A;
		case 'C':
			return C;
		case 'G':
			return G;
		case 'T':
			return T;
		case '$': //因為read都沒有包含$所以這邊就不管回傳什麼了
			return DOLLAR;
		default: return -1;
	}
}

inline __host__ __device__ Type  my_atoi(const char *str) {
	Type ret;
	unsigned char tmp = 0;

	if (str[0] == 'C') {
		tmp = 1;
	} else if (str[0] == 'G') {
		tmp = 2;
	} else if (str[0] == 'T') {
		tmp = 3;
	} else {
		tmp = 0;
	}

	ret = tmp;

	for (int i = 1; i < ENCODELEN; i++) {
		ret = ret << ENCODEBITS;

		if (str[i] == 'C') {
			tmp = 1;
		} else if (str[i] == 'G') {
			tmp = 2;
		} else if (str[i] == 'T') {
			tmp = 3;
        } else if ( str[i] == '\0' ) { // this condition change the spec
            tmp = 0;
            break;
		} else {
			tmp = 0;
		}

		ret = ret | tmp;
	}

	return ret;
}

inline __host__ __device__ char ch_encoding(const char ch) {
	switch(ch) {
		case 'A':
			return '0';
		case 'C':
			return '1';
		case 'G':
			return '2';
		case 'T':
			return '3';
		default:
			printf("So wrong %c!!!\n", ch);
			return '0';
	}
}

__global__ void seq_encoding(char *dna_seq, const int max_task) {
	int globalID = blockIdx.x * blockDim.x + threadIdx.x;
	if (globalID >= max_task) return;

	char *seq = dna_seq + (globalID * LEN_HANDLE_CHAR);
	for (int i = 0; i < LEN_HANDLE_CHAR; i++) {
		seq[i] = ch_encoding(seq[i]);
	}
}

__global__ void bseq_encoding(char *dna_seq, Type *bseq_ptr, const int bseq_size, const size_t segment_offset) {
	const int startID = (blockIdx.y*gridDim.x+blockIdx.x)*(blockDim.x*blockDim.y);
	int globalID = segment_offset + startID + (threadIdx.y*blockDim.x+threadIdx.x);
	if (globalID >= bseq_size) return;

	bseq_ptr[globalID] = my_atoi(dna_seq + (globalID * ENCODELEN));
	printf("gid %d is %u\n", globalID, bseq_ptr[globalID]);
	printf("gid %d offset %d\n", globalID, globalID * ENCODELEN);
}
__global__ void bseq_encoding_check_seq (char *dna_seq, Type *bseq_ptr, const int64_t seq_size, const int64_t segment_offset) {
	const int64_t startID = (blockIdx.y*gridDim.x+blockIdx.x)*(blockDim.x*blockDim.y);
	int64_t globalID = startID + (threadIdx.y*blockDim.x+threadIdx.x);
	if (globalID >= seq_size) return;
    char* const tmp = dna_seq + (globalID * ENCODELEN);
	bseq_ptr[globalID + segment_offset] = my_atoi ( tmp );
	// printf("gid %d is %u\n", globalID, bseq_ptr[globalID + segment_offset]);
	// printf("gid %d offset %d\n", globalID, globalID * ENCODELEN);
}

inline __host__ __device__ int get_occ_table(int position, int token, int occ_table_reduce[][4], char *sbwt_string) {
	int last_record = position / TBLSTRIDE;
	//char tail_char;
	int token_counter;

	token_counter = occ_table_reduce[last_record][token]; 

	char key_token;

	if (token == 0) key_token = 'A';
	else if (token == 1) key_token = 'C';
	else if (token == 2) key_token = 'G';
	else if (token == 3) key_token = 'T';
	else printf("Wrong key token!!!\n");

	last_record = last_record * TBLSTRIDE;
	for (int i = last_record + 1; i <= position; i++) {
		//tail_char = sbwt_string[i - 1];
		//if (tail_char == key_token) token_counter++;
		if (sbwt_string[i - 1] == key_token) token_counter++;
	}

	return token_counter;
}


inline __host__ __device__ int match_reduce(int token, int position, int c_table[], int occ_table_reduce[][4], char *sbwt_string) {
	return c_table[token] + get_occ_table(position, token, occ_table_reduce, sbwt_string);
}

inline __host__ __device__ int binary_search(int search, int data[], int len_data) {
	int low = 0;
	int high = len_data - 1;
	int mid;

	while (low <= high) {
		mid = (low + high) / 2;

		if (data[mid] == search) {
			return mid;
		} else if (search < data[mid]) {
			high = mid - 1;
		} else if (search > data[mid]) {
			low = mid + 1;
		}	
	}

	return -1;
}


inline void __host__ __device__ find_possible_nr3(char *read, int upper, int bottom,  int c_table[], int occ_table_reduce[][4], int len_location_table, int location_table_key[], int location_table_val[], char *sbwt_string, int maxsize, int &resultcnt, const int &resultbase, int resultz[], bool result_rc[], int result_it[], char fbwt_loc_mark[], PositionChecker *poschk, int strand_opt, const int len_read) {
	int his_upper[TBLSTRIDE];
	int his_bottom[TBLSTRIDE];
	int his_token[TBLSTRIDE];

	const int r = TBLSTRIDE - 1;

	his_upper[0] = upper;
	his_bottom[0] = bottom;
	his_token[1] = 0;

	int ret;
	int lowerIter = 0;
	int position;
	bool isRC;
	for (int level = 1; level <= r; ) {
		while (his_token[level] <= T) {
			if (resultcnt == maxsize) return;
			his_upper[level] = match_reduce(his_token[level], his_upper[level-1], c_table, occ_table_reduce, sbwt_string);
			his_bottom[level] = match_reduce(his_token[level], his_bottom[level-1], c_table, occ_table_reduce, sbwt_string);

			bool go_down = true;

			if (his_upper[level] >= his_bottom[level]) go_down = false;
			else {
				for (int x = his_upper[level]; x < his_bottom[level]; x++) {
					if (fbwt_loc_mark[x] == 0) continue;
					if ((ret = binary_search(x, location_table_key, len_location_table)) != -1) {
						position = location_table_val[ret] + level;
						if (poschk->is_vaild(len_read, lowerIter, position, isRC, strand_opt))
						{
							const int idx_result = resultbase + resultcnt;
							//resultz[idx_result] = location_table_val[ret];
							resultz[idx_result] = position;
							result_rc[idx_result] = isRC;
							result_it[idx_result] = lowerIter;
							resultcnt++;
						}
						if (his_bottom[level] - his_upper[level] == 1) go_down = false;
					}
				}
			}

			if (go_down) { // down
				if (level == r) { // end-boundary continue
					his_token[level]++;
					continue;
				}
				his_token[++level] = 0;
				break;
			} else { // same level continue
				his_token[level]++;
			}
		}
		if (resultcnt >= maxsize) return;
		// i >= np
		// next level start
		if (his_token[level] == 0) continue;
		// up
		if (level == 1) return;
		his_token[--level]++;
	}
}

__global__ void find_read(
		char *reads,
		int c_table[],
		int occ_table_reduce[][4],
		int len_occ_table_reduce,
		char *sbwt_string,
		int numSuffix,
		int len_location_table,
		int location_table_key[],
		int location_table_val[],
		int result[],
		int resultz[],
		bool result_rc[],
		int result_it[],
		int max_threads,
		char fbwt_loc_mark[],
		int read_length,
		PositionChecker *poschk,
		int strand_opt
) {
	int globalID = blockIdx.x * blockDim.x + threadIdx.x;
	if(globalID >= max_threads) return;

	char *read = (reads + (globalID * read_length));
	//printf("findqq: %s\n", read);

	const int len_read = read_length - 1;

	int token = get_token_id(&read[len_read - 1]);

	int upper = c_table[token];
	int bottom = c_table[token + 1];

	for (int i = len_read - 2; i >= 0; i--) {
		token = get_token_id(&read[i]);

		upper = match_reduce(token, upper, c_table, occ_table_reduce, sbwt_string);
		bottom = match_reduce(token, bottom, c_table, occ_table_reduce, sbwt_string);
		if (upper >= bottom) {
			result[globalID] = 0;
			return;
		}
	}

	if ((bottom - upper) > OVERNUM) {
		result[globalID] = 0;
		return;
	}

	result[globalID] = bottom - upper;

	int ret;
	int resultcnt = 0;
	int resultbase = globalID * OVERNUM;
	int lowerIter = 0;
	int position;
	bool isRC;
	for (int i = upper; i < bottom; i++) {
		if (fbwt_loc_mark[i] == 0) continue;
		if ((ret = binary_search(i, location_table_key, len_location_table)) != -1) {
			position = location_table_val[ret];
			if (poschk->is_vaild(len_read, lowerIter, position, isRC, strand_opt))
			{
				const int idx_result = resultbase + resultcnt;
				//resultz[idx_result] = location_table_val[ret];
				resultz[idx_result] = position;
				result_rc[idx_result] = isRC;
				result_it[idx_result] = lowerIter;
		  		resultcnt++;
		  	}
			if (bottom - upper == 1) break;
		}
	}

	find_possible_nr3(read, upper, bottom, c_table, (int (*)[4])occ_table_reduce, len_location_table, location_table_key, location_table_val, sbwt_string, (bottom - upper), resultcnt, resultbase, resultz, result_rc, result_it, fbwt_loc_mark, poschk, strand_opt, len_read);

	result[globalID] = resultcnt;
}


inline __host__ __device__ Type  bseq_segment(const int ref_pos, Type *bseq, const int bseq_size) {
	const int bseq_pos = ref_pos / ENCODELEN;
	const int bseq_offset = ref_pos % ENCODELEN;
	Type s1, s2;

	s1 = bseq[bseq_pos];
	s2 = bseq[bseq_pos + 1];

	const int bit_offset = (bseq_offset * ENCODEBITS); 
	s1 = s1 << bit_offset;
	s2 = s2 >> (TYPEBITS - bit_offset);

	return s1 | s2;
}


__global__ void cpy_in_cuda_bseq(Type *key, Type *bseq, int *idx, int round, int max, const int bseq_size) {
	const int startID = (blockIdx.y*gridDim.x+blockIdx.x)*(blockDim.x*blockDim.y);
	int globalID = startID+(threadIdx.y*blockDim.x+threadIdx.x);
	if(globalID >= max) return;

	const int ref_pos = idx[globalID] + round * ENCODELEN;
	key[globalID] = bseq_segment(ref_pos, bseq, bseq_size);
}


inline void simple_cuda_config(dim3 &dim_grid, dim3 &dim_block, int total, int max_thd = MAX_DIM_THD) {
	dim_block.y = 1;
	dim_grid.y = 1;

	dim_block.x = total;
	dim_grid.x = 1;
	if (dim_block.x > max_thd) {
		dim_grid.x = (dim_block.x + max_thd - 1) / max_thd;
		dim_block.x = max_thd;
	}
}

inline void advance_cuda_config(dim3 &dim_grid, dim3 &dim_block, int total, int max_thd = 16, int max_blk = 256) {
	dim_block.x = max_thd;
	dim_block.y = max_thd;
	
	dim_grid.x = max_blk;
	
	const int count = max_blk * max_thd * max_thd;
	dim_grid.y = (total + count - 1) / count;
}


__global__ void sbwt_chain(int suffix_sorted[],char reference[],  char sbwt_string[], int num_suffix) {
	const int startID = (blockIdx.y*gridDim.x+blockIdx.x)*(blockDim.x*blockDim.y);
	int globalID = startID+(threadIdx.y*blockDim.x+threadIdx.x);

	if (globalID >= num_suffix) return;
	
	if (suffix_sorted[globalID] == 0)
		sbwt_string[globalID] = '$';
	else
		sbwt_string[globalID] = *(reference + suffix_sorted[globalID] - 1);
}


////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// api funciton
//
///////////////////////////////////////////////////////////////////////////////////////////////////////
void sbwt_string_create(char *sbwt_string, char *d_reference_char, thrust::host_vector<int> &suffix_array)
{
	dim3 dim3_grid, dim3_block;

	char *d_sbwt_string;
	int num_suffix = suffix_array.size();
	thrust::device_vector<int> d_vals(suffix_array);
	int *d_vals_ptr = thrust::raw_pointer_cast(&d_vals[0]);

	CudaSafeCall( hipMalloc((void **)&d_sbwt_string, sizeof(char) * num_suffix) );

	advance_cuda_config(dim3_grid, dim3_block, num_suffix);

	sbwt_chain<<<dim3_grid, dim3_block>>>(d_vals_ptr, d_reference_char, d_sbwt_string, num_suffix);
	CudaSafeCall( hipMemcpy(sbwt_string, d_sbwt_string, sizeof(char) * num_suffix, hipMemcpyDeviceToHost) );
	hipFree(d_sbwt_string);
}
// d_reference_char, bseq_ptr和bseq_size都是回傳用的
void compress_reference_check(const char *reference_char, Type **bseq_ptr, int &bseq_size)
{
    const char* ref_end ( reference_char + strlen(reference_char) );
    std::vector<Type> bseq_result;
    std::vector<Type> bseq_answer;
    bseq_result.reserve(bseq_size);
    bseq_answer.reserve(bseq_size);
	CudaSafeCall( hipMemcpy( bseq_result.begin().base(), *bseq_ptr, bseq_size * sizeof(Type), hipMemcpyDeviceToHost ) );
	CudaCheckError();
    int k = 0;
    for(const char* itr = reference_char; itr < ref_end; itr += ENCODELEN )
    {
        bseq_answer.push_back(my_atoi(itr));
        k ++;
    }
    // for(size_t i = 0; i < (strlen(reference_char) + ENCODELEN - 1 )/ ENCODELEN; i++)
    // {
    //     bseq_answer[i] = my_atoi(reference_char + (i * ENCODELEN));
    // }
    for(size_t i = 0; i < bseq_size; i ++ )
    {
        assert(bseq_result[i] == bseq_answer[i]);
    }
}
void compress_reference(const char *reference_char, char **d_reference_char, Type **bseq_ptr, int &bseq_size)
{
	const int64_t len_dna_padded = strlen(reference_char);
    std::cout << "len_dna_padded : " << len_dna_padded << std::endl;
    int64_t ref_total_need_mem = sizeof(char) * len_dna_padded + 1;
    std::cout << "ref_total_need_mem : " << ref_total_need_mem << std::endl;
    const int64_t mem_limit = getCudaFreeMemSize() * 0.6 ; // 60% for reference uncompressed data and round to 4's mul
    std::cout << "mem_limit : " << mem_limit << std::endl;
    const int64_t mem_limit_in_char =  ( mem_limit / sizeof(char) ) / ENCODELEN * ENCODELEN; 
    std::cout << "mem_limit_in_char : " << mem_limit_in_char << std::endl;
    int64_t aggrigate_char_num = 0;
	bseq_size = ( len_dna_padded + ENCODELEN - 1) / ENCODELEN;
    std::cout << "bseq_size : " << bseq_size << std::endl;
    std::cout << "ENCODELEN : " << ENCODELEN << std::endl;
	static thrust::device_vector<Type> d_bseq(bseq_size + 1);
	d_bseq[bseq_size] = 0;
	*bseq_ptr = thrust::raw_pointer_cast(&d_bseq[0]);
	// CudaSafeCall( hipMalloc((void **)d_reference_char, sizeof(char) * len_dna_padded) );
	CudaSafeCall( hipMalloc( (void **)d_reference_char, std::min ( 
        (int64_t)(mem_limit_in_char * sizeof(char))
        , ref_total_need_mem 
    ) ) );
	CudaCheckError();
    size_t aggrigate_ref_offset = 0;
    int compress_segment_count(0);
    for( int64_t remain(len_dna_padded) ; remain > 0; remain -= mem_limit_in_char )
    {
	    // CudaSafeCall( hipMemcpy(*d_reference_char, reference_char, sizeof(char) * len_dna_padded, hipMemcpyHostToDevice) );
        std::cout << "aggrigate_ref_offset : " << aggrigate_ref_offset << std::endl;
        std::cout << "copy size : " << std::min ( 
            (int64_t)(mem_limit_in_char * sizeof(char))
            , remain + 1 )  // + 1 for \0
            << std::endl;
        // std::cout << __FILE__ << __LINE__ << reference_char + aggrigate_ref_offset << std::endl;
	    CudaSafeCall( hipMemcpy( 
            *d_reference_char
            , reference_char + aggrigate_ref_offset
            , std::min ( 
                (int64_t)(mem_limit_in_char * sizeof(char))
                , remain + 1 ) // + 1 for \0
            , hipMemcpyHostToDevice) );

	    // bseq is d_reference_char_binary

	    dim3 dim3_grid, dim3_block;

	    // advance_cuda_config(dim3_grid, dim3_block, mem_limit_in_char);
        // size_t bseq_seg_size ( ( mem_limit_in_char + ENCODELEN - 1) / ENCODELEN );
        size_t bseq_seg_size ( 
            std::min(
                mem_limit_in_char / ENCODELEN
                , ( remain + ENCODELEN - 1 )/ ENCODELEN));
        size_t seq_seg_size ( std::min(mem_limit_in_char, remain) );
        std::cout << "seq_seg_size : " << seq_seg_size << std::endl;
        std::cout << "bseq_seg_size : " << bseq_seg_size << std::endl;
	    advance_cuda_config(dim3_grid, dim3_block, bseq_seg_size);
        std::cout << "aggrigate_char_num : " << aggrigate_char_num << std::endl;
	    bseq_encoding_check_seq<<<dim3_grid, dim3_block>>> ( 
            *d_reference_char
            , *bseq_ptr
            // , seq_seg_size
            , bseq_seg_size
            , aggrigate_char_num
        );
        // release d_reference_char
	    CudaCheckError();

        aggrigate_char_num += bseq_seg_size;
        aggrigate_ref_offset += mem_limit_in_char;
        compress_segment_count ++;
    }
    std::cout << "aggrigate_char_num : " << aggrigate_char_num << std::endl;
    std::cout << "compress_segment_count : " << compress_segment_count << '\n';
    CudaSafeCall(hipFree( *d_reference_char ));
}


// suffix_array 要先放入要排的suffix
thrust::host_vector<int> suffix_sort(int sort_length, thrust::host_vector<int> &suffix_array, Type *bseq_ptr, int bseq_size)
{
	dim3 dim3_grid, dim3_block;

	const int num_suffix = suffix_array.size();

	thrust::host_vector<int> &h_vals = suffix_array;
	{
		thrust::device_vector<Type> d_keys(num_suffix);
		thrust::device_vector<int> d_vals;

		if (SUFFIXLEN % ENCODELEN != 0)
			std::cerr << "Warning... mismatch suffixlen\n";

		d_vals = h_vals;

		Type *d_keys_ptr = thrust::raw_pointer_cast(&d_keys[0]);
		int *d_vals_ptr = thrust::raw_pointer_cast(&d_vals[0]);

		advance_cuda_config(dim3_grid, dim3_block, num_suffix);
		//std::cerr << "num of be sorted suffix: " << num_suffix << "\n";

		// 由LSB排到MSB
		const int PART = (sort_length + ENCODELEN - 1)/ ENCODELEN;
		for (int round = PART - 1; round >= 0; round--) {

			// 複製壓縮序列到排序的key陣列
			cpy_in_cuda_bseq<<<dim3_grid, dim3_block>>>(d_keys_ptr, bseq_ptr, d_vals_ptr, round, num_suffix, bseq_size);
			thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_vals.begin(), thrust::less<Type>());
		}

		h_vals = d_vals;
		showCudaUsage();
		CudaCheckError();

	}

	return h_vals;
}

////////////////////////////////////////////////////////////////////////


std::vector<int> make_random_table(int num_suffix, int random_num)
{
	srand(time(NULL));

	std::set<int> random_table;

	for (int i = 0; i < random_num; i++)
	{
		random_table.insert(rand() % num_suffix);
	}

	std::vector<int> random_table_uniq(random_table.begin(), random_table.end());

	return random_table_uniq;
}


void make_split_table(std::vector<int> &split_table, int split_num, const thrust::host_vector<int> &random_table, std::vector<std::string> &archive_name)
{
	// Sampling
	for (int i = 1; i < split_num; i++)
	{
		int idx = random_table.size() / split_num * i;
		split_table.push_back(random_table[idx]);	
	}
	
	for (int i = 0; i < split_num; i++)
	{
		archive_name.push_back( std::string("split_")+boost::lexical_cast<std::string>(i) );
	}
}


__device__
int bseq_cmp(int pos_a, int pos_b, Type *bseq, int bseq_size)
{
	// SUFFIXLEN必須為(TYPEBYTES * ENCODERATIO)的倍數
	const int len_cmp = (SUFFIXLEN + (TYPEBYTES * ENCODERATIO) - 1) / (TYPEBYTES * ENCODERATIO);

	Type a, b;
	for (int i = 0; i < len_cmp; i++)
	{
		a = bseq_segment(pos_a, bseq, bseq_size);
		b = bseq_segment(pos_b, bseq, bseq_size);
		if (a == b) continue;
		else
		{
			if (a > b) return 1;
			else return -1;
		}
	}
	return 0;
}


__device__
int bseq_lower_bound(int search, int data[], int len_data, Type *bseq, int bseq_size) // data is d_sampler_ptr
{
	int first = 0, last = len_data;
	int count = last - first, step;
	int it;

	while (count > 0)
	{
		it = first;
		step = count / 2;
		it += step;
		if (bseq_cmp(search, data[it], bseq, bseq_size) > 0)
		{
			first = ++it;
			count -= step + 1;
		}
		else
		{
			count = step;
		}
	}
	return first;
}


__global__
void classify_seq_tables_cuda(
      int *d_suffixs_ptr
    , int size_suffix
    , int *d_result_ptr
    , int *d_sampler_ptr
    , int size_sampler
    , Type *bseq_ptr
    , int bseq_size
    , const int start_of_this_stage)
{
	const int startID = (blockIdx.y*gridDim.x+blockIdx.x)*(blockDim.x*blockDim.y);
	const int globalID = startID+(threadIdx.y*blockDim.x+threadIdx.x);
	const int accmulated_globalID = globalID + start_of_this_stage;

	if(accmulated_globalID >= size_suffix) return;

	d_result_ptr[globalID] = bseq_lower_bound(d_suffixs_ptr[globalID],  d_sampler_ptr, size_sampler, bseq_ptr, bseq_size);
}


void classify_seq_tables(
      std::vector<int> &split_table
    , Type *bseq_ptr, int bseq_size
    , thrust::host_vector<int> &suffix_array
    // , std::vector< thrust::host_vector<int> > &SeqTables
    , std::vector< std::pair< std::string, uint64_t > >& SeqTables
    , std::vector<std::string> archive_name)
{
    std::vector< std::ofstream* > SeqTablesf;
    SeqTablesf.reserve(SeqTables.size());
    for ( int i = 0; i < SeqTables.size(); i ++ )
    {
        std::pair<std::string, uint64_t>& p = SeqTables[i];
        std::cout << "open file : " << p.first << std::endl;
        SeqTablesf.push_back( new std::ofstream(p.first.c_str()) );
        p.second = 0;
    }
	dim3 dim3_grid, dim3_block;

	const int num_suffix = suffix_array.size();

	thrust::host_vector<int> &h_suffixs = suffix_array;
	thrust::host_vector<int> h_sampler(split_table);


	thrust::device_vector<int> d_sampler(h_sampler);
	int *d_sampler_ptr = thrust::raw_pointer_cast(&d_sampler[0]);
	//showCudaUsage();

	// can not send data over cuda total memory size
	const int64_t MAX_NUM_TO_CUDA = 200000000;

	int64_t start, end = 0;

	for (int i = 0; end < h_suffixs.size(); i++)
	{
		{
			Timer tm("Classify each");
			
			start = i * MAX_NUM_TO_CUDA;
			end = (i + 1) * MAX_NUM_TO_CUDA;
			if (end > h_suffixs.size())
				end = h_suffixs.size();

			advance_cuda_config(dim3_grid, dim3_block, MAX_NUM_TO_CUDA);

			thrust::device_vector<int> d_suffixs((h_suffixs.begin() + start), (h_suffixs.begin() + end));
			//showCudaUsage();
			thrust::device_vector<int> d_result(end - start);
			//showCudaUsage();

			int *d_suffixs_ptr = thrust::raw_pointer_cast(&d_suffixs[0]);
			int *d_result_ptr = thrust::raw_pointer_cast(&d_result[0]);

			{
				Timer in("Classify inner");
				classify_seq_tables_cuda<<<dim3_grid, dim3_block>>>(
                      d_suffixs_ptr
                    , h_suffixs.size()
                    , d_result_ptr
                    , d_sampler_ptr
                    , h_sampler.size()
                    , bseq_ptr
                    , bseq_size
                    , start
                );
			}

			thrust::host_vector<int> h_result = d_result;

			for (int i = 0; i < h_result.size(); i++)
			{
				// SeqTables[h_result[i]].push_back(h_suffixs[i + start]);
				*SeqTablesf[h_result[i]] << h_suffixs[i + start] << '\n';
                // std::cout << h_suffixs[i + start] << std::endl;
                SeqTables[h_result[i]].second ++ ;
			}
		}
	}
    for ( int i = 0; i < SeqTablesf.size(); i ++ )
    {
        SeqTablesf[i]->flush();
        SeqTablesf[i]->close();
        delete SeqTablesf[i];
    }
    
}

void mkq_sort(
      std::vector<std::string> &archive_name
    , thrust::host_vector<int> &suffix_array
    // , std::vector< thrust::host_vector<int> > &SeqTables
    , std::vector< std::pair< std::string, uint64_t > > &SeqTables
    , Type *bseq_ptr
    , int bseq_size
)
{
	for (int i = 0; i < SeqTables.size(); i++)
	{
		//std::cerr << "uuu\n";
		// thrust::host_vector<int> &sub_suffix_array = SeqTables[i];
		thrust::host_vector<int> sub_suffix_array;
        sub_suffix_array.reserve(SeqTables[i].second );
        std::string seq_table_input_line;
        std::ifstream f(SeqTables[i].first.c_str());
        while(std::getline(f, seq_table_input_line))
        {
            sub_suffix_array.push_back(atoi(seq_table_input_line.c_str()));
        }
        f.close(); std::remove(SeqTables[i].first.c_str());
		if (sub_suffix_array.size() != 0) 
		{
			sub_suffix_array = suffix_sort(SUFFIXLEN, sub_suffix_array, bseq_ptr, bseq_size);
		}

		showCudaUsage();
		{
			Timer tt("Split archive");
			// std::vector<int> each_group(SeqTables[i].begin(), SeqTables[i].end());
			std::vector<int> each_group(sub_suffix_array.begin(), sub_suffix_array.end());
			archive_save(archive_name[i], each_group);

			//thrust::host_vector<int> empty;
			//empty.swap(SeqTables[i]);
		}
	}
		
}


void split_sort(
      std::vector<std::string> &archive_name
    , Type *bseq_ptr
    , int64_t bseq_size
    , int64_t num_suffix
    , thrust::host_vector<int> &suffix_array
    , int64_t average_size = 100000000
    , int len_compare = SUFFIXLEN
)
{
	int64_t split_num = num_suffix / average_size;
	if (split_num == 0) split_num = 1;
	std::cerr << "split_num: " << split_num << "\n";
	int64_t random_num = split_num * 4;

	// split_table is used for sampling
	std::vector<int> split_table;
	// std::vector< thrust::host_vector<int> > SeqTables(split_num);
	std::vector< std::pair<std::string, uint64_t> > SeqTables(split_num);
    for( int i = 0; i < split_num; i ++ )
    {
        SeqTables[i].first = "seq_table_" + boost::lexical_cast<std::string>(i);
    }

	std::vector<int> random_table = make_random_table(num_suffix, random_num);
	//std::vector<int> random_sort_w_suffix = suffix_sort(len_compare, random_table, bseq_ptr, bseq_size);
	thrust::host_vector<int> h_random_table(random_table.begin(), random_table.end());
	thrust::host_vector<int> random_sort_w_suffix = suffix_sort(len_compare, h_random_table, bseq_ptr, bseq_size);

	{
		Timer tm("Make split table");
		make_split_table(split_table, split_num, random_sort_w_suffix, archive_name);
	}

	{
		Timer tm("Classify");
		classify_seq_tables(split_table, bseq_ptr, bseq_size, suffix_array, SeqTables, archive_name);
	}

	{
		thrust::host_vector<int> &tmp = suffix_array;
		thrust::host_vector<int> empty;
		empty.swap(suffix_array);
	}


	thrust::host_vector<int> new_suffix_array;
	{
		Timer tm("MKQ sort");
		mkq_sort(archive_name, new_suffix_array, SeqTables, bseq_ptr, bseq_size);
	}

	archive_save("archive_name.archive", archive_name);
}


// input: chrStartPosFile
// output: chr_start_pos
void readChrStartPos (const std::string& chrStartPosFile, std::map<INTTYPE, std::string> &chr_start_pos) {
	std::ifstream in(chrStartPosFile.c_str());
	std::string line, chr;
	INTTYPE startPos = 0;
	while (getline (in, line)) {
		std::stringstream ss(line);
		ss >> chr >> startPos;
		chr_start_pos.insert(std::make_pair (startPos, chr));
	}
}


// input: chrLenFile
// output: _realSize, chr_length
void readChrLen (const std::string& chrLenFile, int &_realSize, std::map<std::string, int> &chr_length) {
	std::ifstream in(chrLenFile.c_str());
	std::string line, chr;
	int length = 0;
	while (getline (in, line)) {
		std::stringstream ss(line);
		ss >> chr >> length;
		// FIXME: replace _realSize...
		_realSize += length;
		if (chr_length.find(chr) == chr_length.end())
			chr_length.insert(std::make_pair (chr, length));
		else {
			std::cerr << "Error: duplicated chromosome name" << std::endl;
			exit (1);
		}
	}
}


// input: fileName
// output: chr_umbiguous_starting_length
void readNPosLen (const std::string& fileName, std::map<INTTYPE, INTTYPE> &chr_umbiguous_starting_length) {
	std::ifstream fp(fileName.c_str(), std::ios::binary);
	boost::archive::binary_iarchive archive_fp(fp);
	archive_fp & chr_umbiguous_starting_length;
	fp.close();
}


template<class Iter>
void PrintCollection(Iter first, Iter last,
		const char* separator="\n",
		const char* arrow="->",
		const char* optcstr="") 
{
	typedef Iter iter_type;
	std::cerr << optcstr;
	for (iter_type begin = first, it = begin, end = last;
			it != end; ++it) {
		if (it != begin) {
			std::cerr << separator;
		}
		std::cerr << it->first << arrow << it->second;
	}
	std::cerr << std::endl;
}


template <class FileType, class StrType>
void fastq_reader(FileType &reads_fs, StrType &read_name, StrType &read_body, StrType &read_opt, StrType &read_quality)
{
	reads_fs >> read_name;
	reads_fs >> read_body;
	reads_fs >> read_opt;
	reads_fs >> read_quality;
}


std::string rc_seq(char *pseq, int len_seq)
{
	std::string seq(pseq, pseq + len_seq);
	std::reverse(seq.begin(), seq.end());
	for (int i = 0; i < seq.length(); i++)
	{
		if (seq[i] == 'A') seq[i] = 'T';
		else if (seq[i] == 'C') seq[i] = 'G';
		else if (seq[i] == 'G') seq[i] = 'C';
		else if (seq[i] == 'T') seq[i] = 'A';
	}

	return seq;
}


inline std::string r_seq(std::string seq)
{
	std::reverse(seq.begin(), seq.end());

	return seq;
}
////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// test funciton
//
///////////////////////////////////////////////////////////////////////////////////////////////////////

__global__
void test_bseq(Type *bseq_ptr, int bseq_size)
{
	for (int i = 0; i < 17; i++)
	{
		printf("test bseq %d %u\n", i, bseq_segment(i, bseq_ptr, bseq_size));
	}
}

__global__
void test_poschk(PositionChecker *poschk)
{
	printf("==[ test poschk ]==\n");
	printf("realSize = %d\n", poschk->_realSize);
	printf("len_chr_start_pos %d\n", poschk->len_chr_start_pos);
	printf("len_NPosLen %d\n", poschk->len_NPosLen);
	for (int i = 0; i < poschk->len_chr_start_pos; i++)
	{
		printf("%lld -> (not needed in cuda anymore)\n", poschk->chr_start_pos_key[i]);
	}

	for (int i = 0; i < poschk->len_NPosLen; i++)
	{
		printf("%lld -> %lld\n", poschk->NPosLen_key[i], poschk->NPosLen_val[i]);
	}
}

void test_split_sort(thrust::host_vector<int> &suffix_array_qq, Type *bseq_ptr, int bseq_size)
{
	const int num_suffix = suffix_array_qq.size();

	std::cerr << "==[ test split sort start]==\n";

	std::vector<std::string> archive_name;
	split_sort(archive_name, bseq_ptr, bseq_size, num_suffix, suffix_array_qq, 40000000);

	std::vector<int> suffix_array;

	for (int i = 0; i < archive_name.size(); i++)
	{
		std::vector<int> sub_suffix_array;
		archive_load(archive_name[i], sub_suffix_array);
		suffix_array.insert(suffix_array.end(), sub_suffix_array.begin(), sub_suffix_array.end());
	}

	showCudaUsage();
	std::cerr << "wwww\n";


	thrust::host_vector<int> suffix_array_golden(num_suffix);
	int from_back = num_suffix - 1;
	for (int i = 0; i < num_suffix; i++) {
		suffix_array_golden[i] = from_back--;
	}
	
	showCudaUsage();
	std::cerr << "wwww\n";

	suffix_array_golden = suffix_sort(SUFFIXLEN, suffix_array_golden, bseq_ptr, bseq_size);

	std::cerr << "suffix_array size " << suffix_array.size() << "\n";
	std::cerr << "suffix_array_golden size " << suffix_array_golden.size() << "\n";

	for (int i = 0; i < suffix_array.size(); i++)
	{
		if (suffix_array[i] != suffix_array_golden[i])
		{
		  std::cerr << i << " " << suffix_array[i] << " " << suffix_array_golden[i] << "\n";
		}
	}
	
	std::cerr << "==[ test split sort finish]==\n";
}


void test_load_archive_my_genome_pre_handler
(
	std::string charStartPosFile, 
	std::string chrLenFile,
	std::string startingLenFile
)
{
	std::map<INTTYPE, std::string> chr_start_pos;
	std::map<std::string, int> chr_length;
	std::map<INTTYPE, INTTYPE> chr_umbiguous_starting_length;
	int _realSize = 0;

	readChrStartPos(charStartPosFile, chr_start_pos);
	readChrLen(chrLenFile, _realSize, chr_length);
	readNPosLen(startingLenFile, chr_umbiguous_starting_length);

	std::cerr << "===[ ChrStartPos ]===\n";
	PrintCollection(chr_start_pos.begin(), chr_start_pos.end());
	std::cerr << "===[ Realsize] ===\n";
	std::cerr << _realSize << "\n";
	std::cerr << "===[ ChrLen ]===\n";
	PrintCollection(chr_length.begin(), chr_length.end());
	std::cerr << "===[ NPosLen ]===\n";
	PrintCollection(chr_umbiguous_starting_length.begin(), chr_umbiguous_starting_length.end());
}



////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// main funciton
//
///////////////////////////////////////////////////////////////////////////////////////////////////////

int STRAND_OPT = FIND_BOTH_STRAND;

void build(int argc, char *argv[])
{
#ifdef IS_CREATE_TBL
	std::string non_reference;
	{
		Timer tm("Pre genome handler");
		my_genome_pre_handler(argv[1], non_reference);
		//std::cerr << non_reference << "\n";
		//test_load_archive_my_genome_pre_handler("my_gph_chrStart", "my_gph_chrLen", "my_gph_NposLen.z");
	}

	dim3 dim3_grid;
	dim3 dim3_block;

	std::string dna;
	dna.swap(non_reference);

	dna.resize(dna.size() - 1);

	char *dna_seq = (char *)dna.c_str();
	const int64_t len_dna_padded = strlen(dna_seq) + SUFFIXLEN;

	const size_t numSuffix = len_dna_padded - (SUFFIXLEN - 1);
	std::cerr << "numSuffix " << numSuffix << "\n";



	char *d_reference_char = NULL;
	Type *bseq_ptr = NULL;
	int bseq_size = 0;

	{
	Timer tm("Table total");
	dna.append(SUFFIXLEN, 'A');

	
	{
		Timer tm("Compress");
		compress_reference(dna.c_str(), &d_reference_char, &bseq_ptr, bseq_size);
        // compress_reference_check(dna.c_str(), &bseq_ptr, bseq_size);
		// test_bseq<<<1, 1>>>(bseq_ptr, bseq_size);
        // exit(1);
	}

	{
		// delete dna
		std::string &tmp = dna;
		std::string tmp2;
		tmp2.swap(tmp);
	}

	thrust::host_vector<int> suffix_array(numSuffix);
	int from_back = numSuffix - 1;
	for (int i = 0; i < numSuffix; i++) {
		suffix_array[i] = from_back--;
	}

//////////////////////////// [ Split ] ///////////////////////////////////////

	//test_split_sort(suffix_array, bseq_ptr, bseq_size);
	//return 0;

	std::vector<std::string> archive_name;
	{
		Timer tm("Split sort total");
		split_sort(archive_name, bseq_ptr, bseq_size, numSuffix, suffix_array, 200000000);
	}


//////////////////////////// [ Split ] ///////////////////////////////////////

	{
		const int len_c_table = 5;
		int *c_table = new int[len_c_table];
		for (int i = 0; i < len_c_table; i++) {
			c_table[i] = 0;
		}
		int len_occ_table_reduce = (numSuffix + TBLSTRIDE -1) / TBLSTRIDE  + 1; // 1 is for ceil
		int *occ_table_reduce = new int[len_occ_table_reduce * 4]; // 4 for 4 char
		int len_location_table = (numSuffix + TBLSTRIDE - 1) / TBLSTRIDE + 1; // ceil and last
		int *location_table_key = new int[len_location_table];
		int *location_table_val = new int[len_location_table];
		int location_cnt = 0;
		int occ_cnt = 0;
		std::vector<char> fbwt_loc_mark_tmp(numSuffix, 0);

		std::string sbwt_string;
		int pitch_accumlate = 0;;
		for (int i = 0; i < archive_name.size(); i++)
		{
			std::vector<int> each_split;
			archive_load(archive_name[i], each_split);
			thrust::host_vector<int> sub_suffix(each_split.begin(), each_split.end());
			char *sub_sbwt = new char[each_split.size() + 1];
			sub_sbwt[each_split.size()] = 0;

			sbwt_string_create(sub_sbwt, d_reference_char, sub_suffix);
			sbwt_string.append(sub_sbwt);

			free(sub_sbwt);

			int pitch_base = pitch_accumlate;
			pitch_accumlate += each_split.size();

			for (int i = 0; i < each_split.size(); i++) {
				if (sub_suffix[i] % TBLSTRIDE == 0) {
					location_table_key[location_cnt] = i + pitch_base;  // key is SBWT index
					location_table_val[location_cnt] = sub_suffix[i]; // val is reference index
					location_cnt++;
					fbwt_loc_mark_tmp[i + pitch_base] = 1;
				}
			}

			for (int i = 0; i < each_split.size(); i++) {
				if (sub_suffix[i] % TBLSTRIDE == 0) {
					fbwt_loc_mark_tmp[i + pitch_base] = 1;
				}
			}


		}

		int xa = 0, xc = 0, xg = 0, xt = 0;
		for (int i = 0; i < numSuffix; i++) {

			if (i % TBLSTRIDE == 0) {
				const int occ_base = occ_cnt * 4;
				occ_table_reduce[occ_base + A] = xa;
				occ_table_reduce[occ_base + C] = xc;
				occ_table_reduce[occ_base + G] = xg;
				occ_table_reduce[occ_base + T] = xt;
				occ_cnt++;
			}

			if (sbwt_string[i] == 'A')
				xa++;
			else if (sbwt_string[i] == 'C')
				xc++;
			else if (sbwt_string[i] == 'G')
				xg++;
			else if (sbwt_string[i] == 'T')
				xt++;

		}

		len_location_table = location_cnt;

		c_table[A] = 1;
		c_table[C] = c_table[A] + xa;
		c_table[G] = c_table[C] + xc;
		c_table[T] = c_table[G] + xg;
		c_table[4] = numSuffix;
	
		///////////////////// Archive save ////////////////////////

		archive_save("sbwt.archive", sbwt_string);

		std::vector<int> occ_table_reduce_v(occ_table_reduce, occ_table_reduce + len_occ_table_reduce * 4);
		archive_save("occ_table.archive", occ_table_reduce_v);

		std::vector<int> c_table_tmp(c_table, c_table + len_c_table);
		archive_save("c_table.archive", c_table_tmp);

		std::vector<int> loc_tbl_key_tmp(location_table_key, location_table_key + len_location_table);
		archive_save("loc_tbl_k.archive", loc_tbl_key_tmp);

		std::vector<int> loc_tbl_val_tmp(location_table_val, location_table_val + len_location_table);
		archive_save("loc_tbl_v.archive", loc_tbl_val_tmp);

		archive_save("fbwt_loc_mark.archive", fbwt_loc_mark_tmp);

		//free(sbwt_string);
		free(occ_table_reduce);
		free(c_table);
		free(location_table_key);
		free(location_table_val);
	}

	}
#endif
}

void map(int argc, char *argv[])
{
#ifdef IS_FIND_READS
	{
		dim3 dim3_grid;
		dim3 dim3_block;

		Timer tmr("Search total");
	///////////////////// Pre genome handler load ////////////////////////

		std::map<INTTYPE, std::string> chr_start_pos;
		std::map<std::string, int> chr_length;
		std::map<INTTYPE, INTTYPE> NPosLen;
		int _realSize = 0;

		readChrStartPos("my_gph_chrStart", chr_start_pos);
		readChrLen("my_gph_chrLen", _realSize, chr_length);
		readNPosLen("my_gph_NposLen.z", NPosLen);

		PositionChecker poschk;
		poschk._realSize = _realSize;
		poschk.len_chr_start_pos = chr_start_pos.size();
		poschk.len_NPosLen = NPosLen.size();
		thrust::device_vector<INTTYPE> dv_chr_start_pos_key(poschk.len_chr_start_pos);
		thrust::device_vector<INTTYPE> dv_NPosLen_key(poschk.len_NPosLen);
		thrust::device_vector<INTTYPE> dv_NPosLen_val(poschk.len_NPosLen);

		std::map<INTTYPE, std::string>::iterator it_chr_start_pos = chr_start_pos.begin();
		for (int i = 0; i < poschk.len_chr_start_pos; i++)
		{
			dv_chr_start_pos_key[i] = it_chr_start_pos->first;
			it_chr_start_pos++;
		}

		std::map<INTTYPE, INTTYPE>::iterator it_NPosLen = NPosLen.begin();
		for (int i = 0; i < poschk.len_NPosLen; i++)
		{
			dv_NPosLen_key[i] = it_NPosLen->first;
			dv_NPosLen_val[i] = it_NPosLen->second;
			it_NPosLen++;
		}

		poschk.chr_start_pos_key = thrust::raw_pointer_cast(dv_chr_start_pos_key.data());
		poschk.NPosLen_key = thrust::raw_pointer_cast(dv_NPosLen_key.data());
		poschk.NPosLen_val = thrust::raw_pointer_cast(dv_NPosLen_val.data());

		PositionChecker *d_poschk;
		CudaSafeCall( hipMalloc((void **)&d_poschk, sizeof(PositionChecker)) );
		CudaSafeCall( hipMemcpy(d_poschk, &poschk, sizeof(PositionChecker), hipMemcpyHostToDevice) );

		//test_poschk<<<1, 1>>>(d_poschk);
		//return 0;

	///////////////////// Archive loc ////////////////////////
		std::string sbwt_arc;
		archive_load("sbwt.archive", sbwt_arc);
		char *p_sbwt_arc = (char *)sbwt_arc.c_str(); // dangerous from const to non-const
		const int sz_sbwt = sbwt_arc.size();
		const int num_suffix = sz_sbwt + 1;

		std::vector<int> occ_table_arc;
		archive_load("occ_table.archive", occ_table_arc);
		int *p_occ_table_arc = &occ_table_arc[0];
		const int sz_occ_table = occ_table_arc.size();

		std::vector<int> c_table_arc;
		archive_load("c_table.archive", c_table_arc);
		int *p_c_table_arc = &c_table_arc[0];
		const int sz_c_table = c_table_arc.size();

		std::vector<int> loc_tbl_key_arc;
		archive_load("loc_tbl_k.archive", loc_tbl_key_arc);
		int *p_loc_tbl_k_arc = &loc_tbl_key_arc[0];
		const int sz_loc_tbl_k = loc_tbl_key_arc.size();

		std::vector<int> loc_tbl_val_arc;
		archive_load("loc_tbl_v.archive", loc_tbl_val_arc);
		int *p_loc_tbl_v_arc = &loc_tbl_val_arc[0];
		const int sz_loc_tbl_v = loc_tbl_val_arc.size();

		std::vector<char> fbwt_loc_mark_arc;
		archive_load("fbwt_loc_mark.archive", fbwt_loc_mark_arc);
		char *p_fbwt_loc_mark_arc = &fbwt_loc_mark_arc[0];
		const int sz_fbwt_loc_mark = fbwt_loc_mark_arc.size();

	/////////////////////         Searching      ////////////////////////

		std::fstream reads_fs;
		reads_fs.open(argv[2]);
		if (!reads_fs.is_open()) {
			std::cerr << "Reads file open failed." << std::endl;
			return;
		}

		std::ofstream out_result(argv[6]);

		int total_num_reads = atoi(argv[5]);

		if (total_num_reads < 1) {
			std::cerr << "Load reads number failed." << std::endl;
			return;
		}


		//////// : transfer to CUDA here
		int *d_result;
		int *d_resultz;
		bool *d_result_rc;
		int *d_result_it;
		char *d_reads;
		int *d_c_table;
		int *d_occ_table_reduce;
		char *d_sbwt_string;
		int *d_location_table_key;
		int *d_location_table_val;
		char *d_fbwt_loc_mark;


		CudaSafeCall( hipMalloc((void **)&d_c_table, sizeof(int) * sz_c_table) );
		CudaSafeCall( hipMemcpy(d_c_table, p_c_table_arc, sizeof(int) * sz_c_table, hipMemcpyHostToDevice) );
		CudaSafeCall( hipMalloc((void **)&d_occ_table_reduce, sizeof(int) * sz_occ_table) );
		CudaSafeCall( hipMemcpy(d_occ_table_reduce, p_occ_table_arc, sizeof(int) * sz_occ_table, hipMemcpyHostToDevice) );
		CudaSafeCall( hipMalloc((void **)&d_sbwt_string, sizeof(char) * (sz_sbwt + 1)) );
		CudaSafeCall( hipMemcpy(d_sbwt_string, p_sbwt_arc, sizeof(char) * (sz_sbwt + 1), hipMemcpyHostToDevice) );
		CudaSafeCall( hipMalloc((void **)&d_location_table_key, sizeof(int) * sz_loc_tbl_k) );
		CudaSafeCall( hipMemcpy(d_location_table_key, p_loc_tbl_k_arc, sizeof(int) * sz_loc_tbl_k, hipMemcpyHostToDevice) );
		CudaSafeCall( hipMalloc((void **)&d_location_table_val, sizeof(int) * sz_loc_tbl_v) );
		CudaSafeCall( hipMemcpy(d_location_table_val, p_loc_tbl_v_arc, sizeof(int) * sz_loc_tbl_v, hipMemcpyHostToDevice) );
		CudaSafeCall( hipMalloc((void **)&d_fbwt_loc_mark, sizeof(char) * sz_fbwt_loc_mark) );
		CudaSafeCall( hipMemcpy(d_fbwt_loc_mark, p_fbwt_loc_mark_arc, sizeof(char) * sz_fbwt_loc_mark, hipMemcpyHostToDevice) );

		int dim_blk = atoi(argv[3]);
		int dim_thd = atoi(argv[4]);

		//showCudaUsage();

		int num_reads = (dim_blk * dim_thd);

		int round = total_num_reads / num_reads;
		if (round == 0)
		{
			num_reads = total_num_reads;
			round = 1;
		}

		std::string peek_read;

		// the second line is read in fastq format
		reads_fs >> peek_read;
		reads_fs >> peek_read;
		const int read_length = peek_read.length() + 1;

		reads_fs.seekg(0);
		reads_fs.seekp(0);

		std::string read_name, read_body, read_opt, read_quality;

		std::vector<std::string> vec_read_name(num_reads);
		std::vector<std::string> vec_read_quality(num_reads);

		for (int y = 0; y < round; y++) {
			//# showCudaUsage();

			char *reads_array = new char[num_reads * read_length];
			char *reads_array_trans;

			int *result;
			int *resultz;
			bool *result_rc;
			int *result_it;
			{
				//@@Timer tt("Allocation");
				for (int i = 0; i < num_reads; i++) {
					reads_array_trans = (reads_array + i * read_length);
					fastq_reader(reads_fs, read_name, read_body, read_opt, read_quality);
					
					vec_read_name[i] = read_name;
					vec_read_quality[i] = read_quality;
					strcpy(reads_array_trans, read_body.c_str());
				}

				CudaSafeCall( hipMalloc((void **)&d_reads, sizeof(char) * num_reads * read_length) );
				CudaSafeCall( hipMemcpy(d_reads, reads_array, sizeof(char) * num_reads * read_length, hipMemcpyHostToDevice) );
				//showCudaUsage();
				CudaSafeCall( hipMalloc((void **)&d_result, sizeof(int) * num_reads) ); // result
				//showCudaUsage();
				CudaSafeCall( hipMalloc((void **)&d_resultz, sizeof(int) * num_reads * OVERNUM) );
				//showCudaUsage();
				CudaSafeCall( hipMalloc((void **)&d_result_rc, sizeof(bool) * num_reads * OVERNUM) );
				//showCudaUsage();
				CudaSafeCall( hipMalloc((void **)&d_result_it, sizeof(int) * num_reads * OVERNUM) );
				showCudaUsage();

				result = new int[num_reads];
				resultz = new int[num_reads * OVERNUM];
				result_rc = new bool[num_reads * OVERNUM];
				result_it = new int[num_reads * OVERNUM];
			}
			{
				//@@Timer tt("Find once");
				find_read<<<dim_blk, dim_thd>>>(d_reads, d_c_table, (int (*)[4])d_occ_table_reduce, (sz_occ_table / 4), d_sbwt_string, num_suffix, sz_loc_tbl_k, d_location_table_key, d_location_table_val, d_result, d_resultz , d_result_rc, d_result_it, num_reads, d_fbwt_loc_mark, read_length, d_poschk, STRAND_OPT);

				CudaCheckError();

				CudaSafeCall( hipMemcpy(result, d_result, sizeof(int) * num_reads, hipMemcpyDeviceToHost) ); 
				CudaSafeCall( hipMemcpy(resultz, d_resultz, sizeof(int) * num_reads * OVERNUM, hipMemcpyDeviceToHost) );
				CudaSafeCall( hipMemcpy(result_rc, d_result_rc, sizeof(bool) * num_reads * OVERNUM, hipMemcpyDeviceToHost) );
				CudaSafeCall( hipMemcpy(result_it, d_result_it, sizeof(int) * num_reads * OVERNUM, hipMemcpyDeviceToHost) );
			}
			{
				//@@Timer tt("IO");
				int base;
				for (int i = 0; i < num_reads; i++) {
					base = i * OVERNUM;
					for (int j = 0; j < result[i]; j++) {
						//std::cout << resultz[base + j] << ' ' << (reads_array + i * read_length) << '\n';
						//std::cout << result_rc[base + j] << '\n';
						//std::cout << result_it[base + j] << '\n';

						std::map<INTTYPE, std::string>::iterator it = chr_start_pos.begin();
						std::advance(it, result_it[base + j]);
						if (result_rc[base + j ] == false)
						{
							std::cout << vec_read_name[i].erase(0, 1) << "\t"
									  << MAPPED << "\t"
									  << it->second << "\t"
									  << resultz[base + j] + 1 << "\t"
									  << 255 << "\t"
									  << (read_length - 1) << "M" << "\t"
									  << "*\t"
									  << 0 << "\t" << 0 << "\t"
									  << (reads_array + i * read_length) << "\t"
									  << vec_read_quality[i] << "\t"
									  << result[i] << "\n";
						}
						else
						{
							std::cout << vec_read_name[i].erase(0, 1) << "\t"
									  << REVERSE_COMPLEMENTED << "\t"
									  << it->second << "\t"
									  << resultz[base + j] + 1 << "\t"
									  << 255 << "\t"
									  << (read_length - 1) << "M" << "\t"
									  << "*\t"
									  << 0 << "\t" << 0 << "\t"
									  << rc_seq(reads_array + i * read_length, read_length - 1) << "\t"
									  << r_seq(vec_read_quality[i]) << "\t"
									  << result[i] << "\n";
						}

					}
				}
			}
			{
				//@@Timer tt("Deallocate");
				CudaSafeCall( hipFree(d_result) );
				CudaSafeCall( hipFree(d_resultz) );
				CudaSafeCall( hipFree(d_result_rc) );
				CudaSafeCall( hipFree(d_result_it) );
				CudaSafeCall( hipFree(d_reads) );
				free(reads_array);
				free(result);
				free(resultz);
				free(result_rc);
				free(result_it);
			}
		}
	}
#endif
}

int main(int argc, char *argv[])
{
	build(argc, argv);
	map(argc, argv);

	return 0;
}


